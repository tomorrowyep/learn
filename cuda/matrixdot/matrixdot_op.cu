﻿#include "hip/hip_runtime.h"


#include <iostream>
#include <random>

using FLOAT = double;
constexpr int g_mRowCount = 1000; // m*n
constexpr int g_nCount = 1000;
constexpr int g_kColCount = 1000; // n*k
constexpr int g_blockSize = 16;

__managed__ FLOAT matrix01[g_mRowCount * g_nCount];
__managed__ FLOAT matrix02[g_nCount * g_kColCount];
__managed__ FLOAT result_cpu[g_mRowCount * g_kColCount];
__managed__ FLOAT result_gpu[g_mRowCount * g_kColCount];

// 原理是将dx，dy分成很多子块加载进入共享内存
__global__ void matrix_gpu(FLOAT* dx, FLOAT* dy)
{
	__shared__ FLOAT step_x[g_blockSize][g_blockSize];
	__shared__ FLOAT step_y[g_blockSize][g_blockSize];

	// 全局索引
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	int index = 0;
	FLOAT sum = 0.;
	for (int step = 0; step <= g_nCount / g_blockSize; ++step)// 分成几个子块累加，一个线程处理结果矩阵中的一个数据
	{
		int step_row = row;// 计算子块的行索引
		int step_col = step * g_blockSize + threadIdx.x; // 计算子块的列索引
		index = step_row * g_nCount + step_col;// dx中的索引
		if (step_row >= g_mRowCount || step_col >= g_nCount)
			step_x[threadIdx.y][threadIdx.x] = 0;
		else
			step_x[threadIdx.y][threadIdx.x] = dx[index];

		step_row = step * g_blockSize + threadIdx.y;
		step_col = col;
		index = step_row * g_kColCount + step_col;// dy中的索引
		if (step_row >= g_nCount || step_col >= g_kColCount)
			step_y[threadIdx.y][threadIdx.x] = 0;
		else
			step_y[threadIdx.y][threadIdx.x] = dy[index];

		__syncthreads();// 同步一下，保证数据写入共享内存了

		for (int i = 0; i < g_blockSize; ++i)
		{
			sum += step_x[threadIdx.y][i] * step_y[i][threadIdx.x];
		}
		__syncthreads();
	}

	if (row < g_mRowCount && col < g_kColCount)
		result_gpu[row * g_kColCount + col] = sum;
}

void matrix_cpu(FLOAT* hx, FLOAT* hy)
{
	for (int row = 0; row < g_mRowCount; ++row)
	{
		for (int col = 0; col < g_kColCount; ++col)
		{
			FLOAT tmp = 0.;
			for (int step = 0; step < g_nCount; ++step)
			{
				tmp += hx[row * g_nCount + step] * hy[step * g_kColCount + col];
			}
			result_cpu[row * g_kColCount + col] = tmp;
		}
	}
}

int main()
{
	// 初始化随机数引擎
	std::random_device rd;
	std::mt19937 gen(rd());
	std::uniform_int_distribution<> dis(1, 100);

	for (int row = 0; row < g_mRowCount; ++row)
	{
		for (int col = 0; col < g_nCount; ++col)
		{
			matrix01[row * g_nCount + col] = dis(gen);
		}
	}

	for (int row = 0; row < g_nCount; ++row)
	{
		for (int col = 0; col < g_kColCount; ++col)
		{
			matrix02[row * g_kColCount + col] = dis(gen);
		}
	}

	int gridRows = (g_mRowCount + g_blockSize - 1) / g_blockSize;
	int gridCols = (g_kColCount + g_blockSize - 1) / g_blockSize;

	dim3 grid(gridCols, gridRows);
	dim3 block(g_blockSize, g_blockSize);

	matrix_cpu(matrix01, matrix02);

	matrix_gpu<<<grid, block>>>(matrix01, matrix02);
	hipDeviceSynchronize();// 等待gpu完成

	// 验证结果
	bool error = true;
	for (int row = 0; row < g_mRowCount; ++row)
	{
		for (int col = 0; col < g_kColCount; ++col)
		{
			if (fabs(result_cpu[row * g_kColCount + col] - result_gpu[row * g_kColCount + col]) > (1.0e-10))
			{
				error = false;
				break;
			}
		}

		if (!error)
			break;
	}

	if (error)
		std::cout << "error = true" << "\n";
	else
		std::cout << "error = false" << "\n";

	return 0;
}