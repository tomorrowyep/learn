#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <random>

using FLOAT = double;
constexpr int g_rowCount = 1000;
constexpr int g_colCount = 1000;
constexpr int g_blockSize = 16;

// 矩阵相乘的关键点就是找到结果矩阵对应元素与相乘矩阵元素之间的关系
// cuda中传入的dim3参数的含义是列行

void hostMatrixDot(FLOAT* hx, FLOAT* hy, FLOAT* hr, int size)
{
    for (int row = 0; row < g_rowCount; ++row)
    {
        for (int col = 0; col < g_colCount; ++col)
        {
            FLOAT sums = 0.;
            for (int step = 0; step < size; ++step)
            {
                sums += hx[row * size + step] * hy[size * step + col];
            }
            hr[row * size + col] = sums;
        }
    }
}

__global__ void matrixDot(FLOAT* dx, FLOAT* dy, FLOAT* dr, int size)
{
    int row = g_blockSize * blockIdx.y + threadIdx.y;
    int col = g_blockSize * blockIdx.x + threadIdx.x;
    FLOAT sums = 0.;
    if (row < size && col < size)
    {
        for (int step = 0; step < size; ++step)
        {
            sums += dx[row * size + step] * dy[size * step + col];
        }
        dr[row * size + col] = sums;
    }
}

int main()
{
    int dataSize = g_rowCount * g_colCount;
    int nbytes = dataSize * sizeof(FLOAT);

    // 定义并分配一维数组
    FLOAT* hx = new FLOAT[dataSize];
    FLOAT* hy = new FLOAT[dataSize];
    FLOAT* hr = new FLOAT[dataSize];
    FLOAT* hrr = new FLOAT[dataSize];
    FLOAT* dx = nullptr;
    FLOAT* dy = nullptr;
    FLOAT* dr = nullptr;

    // 初始化随机数引擎
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_int_distribution<> dis(1, 100);

    // 初始化矩阵乘积的数组，以行主序方式填充一维数组
    for (int row = 0; row < g_rowCount; ++row)
    {
        for (int col = 0; col < g_colCount; ++col) 
        {
            hx[row * g_colCount + col] = dis(gen);
            hy[row * g_colCount + col] = dis(gen);
        }
    }

    // 在设备上分配一维数组
    hipMalloc((void**)&dx, nbytes);
    hipMalloc((void**)&dy, nbytes);
    hipMalloc((void**)&dr, nbytes);
    if (!dx || !dy || !dr)
        return 0;

    // 将主机数据传输到设备
    hipMemcpy(dx, hx, nbytes, hipMemcpyHostToDevice);
    hipMemcpy(dy, hy, nbytes, hipMemcpyHostToDevice);

    // 阻塞，直至gpu返回
    hipDeviceSynchronize();

    int gridRows = (g_rowCount + g_blockSize - 1) / g_blockSize;
    int gridCols = (g_colCount + g_blockSize - 1) / g_blockSize;

    dim3 grid(gridCols, gridRows);
    dim3 block(g_blockSize, g_blockSize);

    matrixDot << <grid, block>> > (dx, dy, dr, g_colCount);

    // 阻塞，直至gpu返回
    hipDeviceSynchronize();

    // 返回结果
    hipMemcpy(hrr, dr, nbytes, hipMemcpyDeviceToHost);

    hostMatrixDot(hx, hy, hr, g_colCount);

    // 验证结果
    bool error = true;
    for (int row = 0; row < g_rowCount; ++row)
    {
        for (int col = 0; col < g_colCount; ++col)
        {
            if (fabs(hr[row * g_colCount + col] - hrr[row * g_colCount + col]) > (1.0e-10))
            {
                error = false;
                break;
            }  
        }

        if (!error)
            break;
    }

    hipFree(dx);
    hipFree(dy);
    hipFree(dr);

    delete[] hx;
    delete[] hy;
    delete[] hr;
    delete[] hrr;

    if (error)
        std::cout << "error = true" << "\n";
    else
        std::cout << "error = false" << "\n";

	return 0;
}
