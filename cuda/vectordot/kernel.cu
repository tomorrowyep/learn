﻿#include <iostream>
#include <random>

#include "hip/hip_runtime.h"


// 适用于一维块和二维网格
#define get_tid() (blockDim.x * (blockIdx.x + blockIdx.y * gridDim.x) + threadIdx.x) // 线程全局索引
#define get_bid() (blockIdx.x + blockIdx.y * gridDim.x) // 块全局索引

using FLOAT = double;

// 每个线程块中线程的数量
constexpr int threadPerBlockNums = 256;

__device__ void  warpReduce(volatile FLOAT* sdata, int tid)
{
    // 同一个线程束一次性计算完，该线程块最终的计算结果为sdata[0]
    sdata[tid] += sdata[tid + 32];
    sdata[tid] += sdata[tid + 16];
    sdata[tid] += sdata[tid + 8];
    sdata[tid] += sdata[tid + 4];
    sdata[tid] += sdata[tid + 2];
    sdata[tid] += sdata[tid + 1];
}

// 每个块处理256个数据，结果存在对应的块索引中
__global__ void dotStage1(FLOAT* dx, FLOAT* dy, FLOAT* dz, int size)
{
    // 每个块处理256个数据
    __shared__ FLOAT sdatas[threadPerBlockNums];

    int globleThreadid = get_tid();
    int blockThreadid = threadIdx.x;
    int blockid = get_bid();

    if (globleThreadid < size)
        sdatas[blockThreadid] = dx[globleThreadid] * dy[globleThreadid];
    else
        sdatas[blockThreadid] = 0.f;

    // 块内同步，保证sdatas已经被当前线程块设置好
    __syncthreads();

    if (blockThreadid < 128)
        sdatas[blockThreadid] += sdatas[blockThreadid + 128];

    __syncthreads();

    if (blockThreadid < 64)
        sdatas[blockThreadid] += sdatas[blockThreadid + 64];

    __syncthreads();

    if (blockThreadid < 32)
        warpReduce(sdatas, blockThreadid);

    __syncthreads();

    if (blockThreadid == 0)
        dz[blockid] = sdatas[0];
}

__global__ void dotStage2(FLOAT* dx, FLOAT* d, int size)
{
    // 每个块处理256个数据
    __shared__ FLOAT sdatas[threadPerBlockNums];

    int globleThreadid = get_tid();
    int blockThreadid = threadIdx.x;
    int blockid = get_bid();

    if (globleThreadid < size)
        sdatas[blockThreadid] = dx[globleThreadid];
    else
        sdatas[blockThreadid] = 0.f;

    // 块内同步，保证sdatas已经被当前线程块设置好
    __syncthreads();

    if (blockThreadid < 128)
        sdatas[blockThreadid] += sdatas[blockThreadid + 128];

    __syncthreads();

    if (blockThreadid < 64)
        sdatas[blockThreadid] += sdatas[blockThreadid + 64];

    __syncthreads();

    if (blockThreadid < 32)
        warpReduce(sdatas, blockThreadid);

    __syncthreads();

    if (blockThreadid == 0)
        d[blockid] = sdatas[0];
}

__global__ void dotStage3(FLOAT* d, int size)
{
    // 每个块处理256个数据
    __shared__ FLOAT sdatas[threadPerBlockNums];
    int blockThreadid = threadIdx.x;

    if (blockThreadid < size)
        sdatas[blockThreadid] = d[blockThreadid];
    else
        sdatas[blockThreadid] = 0.f;

    __syncthreads();

    if (blockThreadid < 128)
        sdatas[blockThreadid] += sdatas[blockThreadid + 128];

    __syncthreads();

    if (blockThreadid < 64)
        sdatas[blockThreadid] += sdatas[blockThreadid + 64];

    __syncthreads();

    if (blockThreadid < 32)
        warpReduce(sdatas, blockThreadid);

    __syncthreads();

    if (blockThreadid == 0)
        d[0] = sdatas[0];
}

void dotProduct(FLOAT* dx, FLOAT* dy, FLOAT* dz, FLOAT* d, int size)
{
    /*第一阶段，完成乘法的计算，转为blockNums大小的数组求和的计算*/
    // 计算二维grid的大小
     // 线程块的数量
    int blockNums = (size + threadPerBlockNums - 1) / threadPerBlockNums;
    int nums = (int)ceil(sqrt(blockNums));
    dim3 grid = dim3(nums, nums);

    dotStage1<<<grid, threadPerBlockNums>>>(dx, dy, dz, size);

    /*第二阶段(可以循环计算的，直到数据量级为一个块的大小)，将blockNums大小的数组长度再减少threadPerBlockNums倍*/
     // 线程块的数量
    int blockNums2 = (blockNums + threadPerBlockNums - 1) / threadPerBlockNums;
    nums = (int)ceil(sqrt(blockNums2));
    grid = dim3(nums, nums);

    dotStage2<<<grid, threadPerBlockNums>>>(dz, d, blockNums);

    /*第三阶段，将blockNums2大小的数组长度再减少threadPerBlockNums倍*/
    dotStage3 <<<1, threadPerBlockNums >>>(d, blockNums2);
}

int main()
{
    int size = 10000070;
    int nbytes = size * sizeof(FLOAT);

    FLOAT* hx = nullptr, * hy = nullptr;
    FLOAT* dx = nullptr, * dy = nullptr, * dz = nullptr, * d = nullptr;

    // 分配cpu的内存
    hx = new FLOAT[size];
    hy = new FLOAT[size];

    // 初始化随机数引擎（1-100）
    std::random_device rd; // 用于生成随机种子
    std::mt19937 gen(rd()); // 随机数引擎
    std::uniform_int_distribution<> dis(1, 100); // 生成 1 到 100 的随机整数

    // 初始化内积的数组
    for (size_t index = 0; index < size; ++index)
    {
        hx[index] = (FLOAT)dis(gen);
        hy[index] = (FLOAT)dis(gen);
    }

    // 分配GPU的内存，加上255是为了不漏掉任何数据
    hipMalloc((void**)&dx, nbytes);
    hipMalloc((void**)&dy, nbytes);
    hipMalloc((void**)&dz, sizeof(FLOAT) * ((size + 255) / 256));
    hipMalloc((void**)&d, sizeof(FLOAT) * ((size + 255) / 256));
    if (!dx || !dy || !dz || !d)
        return 0;

    // 传输数据
    hipMemcpy(dx, hx, nbytes, hipMemcpyHostToDevice);
    hipMemcpy(dy, hy, nbytes, hipMemcpyHostToDevice);

    // 阻塞，直至gpu返回
    hipDeviceSynchronize();

    // 调用核函数
    dotProduct(dx, dy, dz, d, size);

    // 阻塞，直至gpu返回
    hipDeviceSynchronize();

    FLOAT result = 0.f;
    hipMemcpy(&result, d, sizeof(FLOAT), hipMemcpyDeviceToHost);

    std::cout << result << "\n";

    // 使用cpu计算一下，验证结果是否正确
    result = 0.f;
    for (int i = 0; i < size; ++i)
    {
        result += hx[i] * hy[i];
    }

    std::cout << result << "\n";
}